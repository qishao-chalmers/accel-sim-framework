#include "hip/hip_runtime.h"
/* Author1: Mahmoud Khairy, abdallm@purdue.com - 2019 */
/* Author2: Jason Shen, shen203@purdue.edu - 2019 */

#include <stdint.h>
#include <stdio.h>

#include <cstdarg>

#include "utils/utils.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the inst_trace_t structure */
#include "common.h"

/* Instrumentation function that we want to inject, please note the use of
 *  extern "C" __device__ __noinline__
 *    To prevent "dead"-code elimination by the compiler.
 */
extern "C" __device__ __noinline__ void
instrument_inst(int pred, int opcode_id, int32_t vpc, bool is_mem,
                uint64_t addr, int32_t width, int32_t desReg, int32_t srcReg1,
                int32_t srcReg2, int32_t srcReg3, int32_t srcReg4,
                int32_t srcReg5, int32_t srcNum, uint64_t immediate,
                uint64_t pchannel_dev, uint64_t ptotal_dynamic_instr_counter,
                uint64_t preported_dynamic_instr_counter, uint64_t pstop_report,
                uint32_t line_num, bool is_store, int32_t store_data_type, 
                int32_t num_store_data_regs, ...) {
  const int active_mask = __ballot_sync(__activemask(), 1);
  const int predicate_mask = __ballot_sync(__activemask(), pred);
  const int laneid = get_laneid();
  const int first_laneid = __ffs(active_mask) - 1;

  if ((*((bool *)pstop_report))) {
    if (first_laneid == laneid) {
      atomicAdd((unsigned long long *)ptotal_dynamic_instr_counter, 1);
      return;
    }
  }

  inst_trace_t ma;

  if (is_mem) {
    /* collect memory address information */
    for (int i = 0; i < 32; i++) {
      ma.addrs[i] = __shfl_sync(active_mask, addr, i);
    }
    ma.width = width;
    ma.is_mem = true;
  } else {
    ma.is_mem = false;
  }

  // Initialize store data fields
  ma.is_store = is_store;
  ma.store_data_type = (store_data_type_t)store_data_type;
  ma.num_store_data_regs = num_store_data_regs;
  
  // Capture store data if this is a store operation
  if (is_store && num_store_data_regs > 0) {
    va_list vl;
    va_start(vl, num_store_data_regs);
    
    for (int reg_idx = 0; reg_idx < num_store_data_regs && reg_idx < MAX_STORE_DATA_REGS; reg_idx++) {
      // nvbit always provides register values as uint32_t
      // We need to interpret them based on the data type
      uint32_t raw_reg_val = va_arg(vl, uint32_t);
      uint64_t store_val = 0;
      
      // Handle different data types appropriately
      switch (ma.store_data_type) {
        case STORE_DATA_FLOAT32: {
          // For float32, the uint32_t contains the correct bit pattern
          store_val = raw_reg_val;
          break;
        }
        case STORE_DATA_FLOAT64: {
          // For float64, we need two consecutive 32-bit registers
          // This is the first register (lower 32 bits)
          store_val = raw_reg_val;
          if (reg_idx + 1 < num_store_data_regs) {
            // Get the next register for upper 32 bits
            uint32_t upper_reg_val = va_arg(vl, uint32_t);
            store_val |= ((uint64_t)upper_reg_val << 32);
            reg_idx++; // Skip the next iteration since we consumed two registers
          }
          break;
        }
        case STORE_DATA_INT64: {
          // For int64, we need two consecutive 32-bit registers
          store_val = raw_reg_val;
          if (reg_idx + 1 < num_store_data_regs) {
            uint32_t upper_reg_val = va_arg(vl, uint32_t);
            store_val |= ((uint64_t)upper_reg_val << 32);
            reg_idx++; // Skip the next iteration
          }
          break;
        }
        case STORE_DATA_INT32:
        case STORE_DATA_INT16:
        case STORE_DATA_INT8:
        default: {
          // For smaller integer types, just use the lower bits
          store_val = raw_reg_val;
          break;
        }
      }
      
      // Collect store data values from all threads in the warp
      for (int tid = 0; tid < 32; tid++) {
        ma.store_data[tid][reg_idx] = __shfl_sync(active_mask, store_val, tid);
      }
    }
    va_end(vl);
  } else {
    // Initialize store data to zero if not a store operation
    for (int tid = 0; tid < 32; tid++) {
      for (int reg_idx = 0; reg_idx < MAX_STORE_DATA_REGS; reg_idx++) {
        ma.store_data[tid][reg_idx] = 0;
      }
    }
  }

  int4 cta = get_ctaid();
  int uniqe_threadId = threadIdx.z * blockDim.y * blockDim.x +
                       threadIdx.y * blockDim.x + threadIdx.x;
  ma.line_num = line_num;
  ma.warpid_tb = uniqe_threadId / 32;

  ma.cta_id_x = cta.x;
  ma.cta_id_y = cta.y;
  ma.cta_id_z = cta.z;
  ma.warpid_sm = get_warpid();
  ma.opcode_id = opcode_id;
  ma.vpc = vpc;
  ma.GPRDst = desReg;
  ma.GPRSrcs[0] = srcReg1;
  ma.GPRSrcs[1] = srcReg2;
  ma.GPRSrcs[2] = srcReg3;
  ma.GPRSrcs[3] = srcReg4;
  ma.GPRSrcs[4] = srcReg5;
  ma.numSrcs = srcNum;
  ma.imm = immediate;
  ma.active_mask = active_mask;
  ma.predicate_mask = predicate_mask;
  ma.sm_id = get_smid();

  /* first active lane pushes information on the channel */
  if (first_laneid == laneid) {
    ChannelDev *channel_dev = (ChannelDev *)pchannel_dev;
    channel_dev->push(&ma, sizeof(inst_trace_t));
    atomicAdd((unsigned long long *)ptotal_dynamic_instr_counter, 1);
    atomicAdd((unsigned long long *)preported_dynamic_instr_counter, 1);
  }
}
