#include "hip/hip_runtime.h"
/* Author1: Mahmoud Khairy, abdallm@purdue.com - 2019 */
/* Author2: Jason Shen, shen203@purdue.edu - 2019 */

#include <assert.h>
#include <inttypes.h>
#include <stdint.h>
#include <stdio.h>
#include <sys/stat.h>
#include <unistd.h>

#include <algorithm>
#include <bitset>
#include <iostream>
#include <iterator>
#include <map>
#include <sstream>
#include <string>
#include <unordered_map>
#include <unordered_set>
#include <vector>
#include <regex>
/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the inst_trace_t structure */
#include "common.h"

#define TRACER_VERSION "5"

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;
int enable_compress = 1;
int print_core_id = 0;
int exclude_pred_off = 1;
int active_from_start = 1;
int lineinfo = 0;
/* used to select region of interest when active from start is 0 */
bool active_region = true;

/* Should we terminate the program once we are done tracing? */
int terminate_after_limit_number_of_kernels_reached = 0;
int user_defined_folders = 0;

/* Use xz to compress the *.trace file */
int xz_compress_trace = 0;

/* Memory dumping control variables */
int enable_memory_dump = 0;
int memory_dump_compress = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;

std::string user_folder = getcwd(NULL, 0);
std::string cwd = getcwd(NULL, 0);
std::string traces_location = cwd + "/traces/";
std::string kernelslist_location = cwd + "/traces/kernelslist";
std::string stats_location = cwd + "/traces/stats.csv";

std::unordered_map<hipCtx_t, std::string> ctx_kernelslist;
std::unordered_map<hipCtx_t, std::string> ctx_stats_location;
std::unordered_map<hipCtx_t, int> ctx_kernelid;
std::unordered_map<hipCtx_t, FILE *> ctx_resultsFile;
std::unordered_map<hipCtx_t, int> ctx_memcpy_counter;

std::string kernel_ranges = "";

struct KernelRange {
  uint64_t start;
  uint64_t end; // UINT64_MAX means open-ended
  std::vector<std::regex> kernel_name_regexes;  // Vector of regexes for multiple patterns
};
std::vector<KernelRange> g_kernel_ranges;
uint64_t g_max_kernel_id = 0;
void parse_kernel_ranges_from_env() {
  g_kernel_ranges.clear();
  g_max_kernel_id = 0;

  const char* env_var = std::getenv("DYNAMIC_KERNEL_RANGE");
  if (!env_var || std::string(env_var).empty()) {
      g_kernel_ranges.push_back({0, 0, {std::regex(".*")}});  // 0 end = trace all
      return;
  }
  std::istringstream iss(env_var);
    std::string token;
    while (iss >> token) {
        size_t dash_pos = token.find('-');
        size_t regex_pos = token.find('@');  // kernel name indicated by @
        uint64_t start = 0;
        uint64_t end = 0;

        if (regex_pos != std::string::npos) {
            // Kernel name range with regex
            std::string range_part = token.substr(0, regex_pos);
            std::string regex_str = token.substr(regex_pos + 1);

          

            // Parse the range part for start and end
            size_t dash_pos_range = range_part.find('-');
            if (dash_pos_range != std::string::npos) {
                start = std::stoull(range_part.substr(0, dash_pos_range));
                end = std::stoull(range_part.substr(dash_pos_range + 1));
            } else {
                start = std::stoull(range_part);
                end = start;
            }

            // Split multiple regexes by commas
            std::vector<std::string> regex_strings;
            std::istringstream regex_stream(regex_str);
            std::string regex_token;
            while (std::getline(regex_stream, regex_token, ',')) {
                try {
                    g_kernel_ranges.push_back({start, end, {std::regex(regex_token)}});
                } catch (const std::regex_error& e) {
                    std::cerr << "Invalid regex: " << regex_token << std::endl;
                }
            }
        } else {
            // Normal range without kernel name regex
            size_t dash_pos_range = token.find('-');

            if (dash_pos_range != std::string::npos) {
                start = std::stoull(token.substr(0, dash_pos_range));
                end = std::stoull(token.substr(dash_pos_range + 1));
            } else {
                start = std::stoull(token);
                end = start;
            }

            g_kernel_ranges.push_back({start, end, {std::regex(".*")}});
        }

        // Update max kernel ID if needed
        if (end > g_max_kernel_id) {
            g_max_kernel_id = end;
        }
    }


}

bool should_trace_kernel(uint64_t kernel_id, const std::string& kernel_name) {
  for (const auto& range : g_kernel_ranges) {
    // Check range for kernel ID
    if (range.end == 0) {
        if (kernel_id >= range.start) {
            // Match any of the regexes for this range
            for (const auto& regex : range.kernel_name_regexes) {
                if (std::regex_match(kernel_name, regex)) {
                    return true;
                }
            }
        }
    } else if (kernel_id >= range.start && kernel_id <= range.end) {
        // Match any of the regexes for this range
        for (const auto& regex : range.kernel_name_regexes) {
            if (std::regex_match(kernel_name, regex)) {
                return true;
            }
        }
    }
  }
  return false;
}


enum address_format { list_all = 0, base_stride = 1, base_delta = 2 };

/* File pointers for the kernels, and stats files */
static FILE *kernelsFile = NULL;
static FILE *statsFile = NULL;
static bool first_call = true;

unsigned old_total_insts = 0;
unsigned old_total_reported_insts = 0;

void nvbit_at_init() {
  setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
  GET_VAR_INT(
      instr_begin_interval, "INSTR_BEGIN", 0,
      "Beginning of the instruction interval where to apply instrumentation");
  GET_VAR_INT(instr_end_interval, "INSTR_END", UINT32_MAX,
              "End of the instruction interval where to apply instrumentation");
  GET_VAR_INT(exclude_pred_off, "EXCLUDE_PRED_OFF", 1,
              "Exclude predicated off instruction from count");
  GET_VAR_INT(lineinfo, "TRACE_LINEINFO", 0,
              "Include source code line info at the start of each traced line. "
              "The target binary must be compiled with -lineinfo or "
              "--generate-line-info");
  GET_VAR_STR(kernel_ranges, "DYNAMIC_KERNEL_RANGE",
  "Specify kernel IDs or ranges to trace. Format:\n"
  "  - Single ID:       \"2\" traces only kernel 2.\n"
  "  - Range:           \"5-8\" traces kernels 5 through 8 (inclusive).\n"
  "  - Open-ended:      \"10-\" traces from kernel 10 onward.\n"
  "  - Multiple ranges: \"2 5-8 10-\" (space-separated).\n"
  "  - With regex:      \"5-8@kernel_a.*,kernel_b.*\" traces kernels 5–8 with matching names.\n"
  "If unset or empty, all kernels will be traced from the beginning.");
GET_VAR_INT(
      active_from_start, "ACTIVE_FROM_START", 1,
      "Start instruction tracing from start or wait for hipProfilerStart "
      "and hipProfilerStop. If set to 0, DYNAMIC_KERNEL_RANGE options have no "
      "effect");
  GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
  GET_VAR_INT(enable_compress, "TOOL_COMPRESS", 1, "Enable traces compression");
  GET_VAR_INT(print_core_id, "TOOL_TRACE_CORE", 0,
              "write the core id in the traces");
  GET_VAR_INT(
      terminate_after_limit_number_of_kernels_reached, "TERMINATE_UPON_LIMIT",
      0, "Stop the process once the current kernel > DYNAMIC_KERNEL_LIMIT_END");
  GET_VAR_INT(user_defined_folders, "USER_DEFINED_FOLDERS", 0,
              "Uses the user defined "
              "folder TRACES_FOLDER path environment");
  GET_VAR_INT(xz_compress_trace, "TRACE_FILE_COMPRESS", 1,
              "Create xz-compressed trace"
              "file");
  GET_VAR_INT(enable_memory_dump, "ENABLE_MEMORY_DUMP", 1,
              "Enable memory dump");
  GET_VAR_INT(memory_dump_compress, "MEMORY_DUMP_COMPRESS", 0,
              "Enable memory dump compression");
  std::string pad(100, '-');
  printf("%s\n", pad.c_str());

  
  active_region = false;
  char * usr_defined_folder = std::getenv("TRACES_FOLDER");
  if (usr_defined_folder != NULL)
    user_folder = usr_defined_folder;
  parse_kernel_ranges_from_env();

}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

/* instrument each memory instruction adding a call to the above instrumentation
 * function */
void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
  std::vector<hipFunction_t> related_functions =
      nvbit_get_related_functions(ctx, func);

  /* add kernel itself to the related function vector */
  related_functions.push_back(func);

  /* iterate on function */
  for (auto f : related_functions) {
    /* "recording" function was instrumented, if set insertion failed
     * we have already encountered this function */
    if (!already_instrumented.insert(f).second) {
      continue;
    }

    const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
    if (verbose) {
      printf("Inspecting function %s at address 0x%lx\n",
             nvbit_get_func_name(ctx, f), nvbit_get_func_addr(ctx,f));
    }

    uint32_t cnt = 0;
    /* iterate on all the static instructions in the function */
    for (auto instr : instrs) {
      uint32_t line_num = 0;
      // Temporary workaround for a bug in NVBit 1.7.4, which does not correctly
      // handle `call.rel`. Instrumenting this instruction leads to illegal
      // memory access. Refer to:
      // https://github.com/NVlabs/NVBit/issues/142#issue-2911561744
      if (!strcmp(instr->getOpcode(), "CALL.REL.NOINC")) {
        printf("Warning: Ignoring CALL.REL.NOINC (NVBit 1.7.4 bug)\n");
        continue;
      }

      if (cnt < instr_begin_interval || cnt >= instr_end_interval) {
        cnt++;
        continue;
      }

      if (verbose) {
        instr->printDecoded();
      }

      if (lineinfo) {
        char *file_name, *dir_name;
        nvbit_get_line_info(ctx, func, instr->getOffset(), &file_name,
                            &dir_name, &line_num);
      }

      if (opcode_to_id_map.find(instr->getOpcode()) == opcode_to_id_map.end()) {
        int opcode_id = opcode_to_id_map.size();
        opcode_to_id_map[instr->getOpcode()] = opcode_id;
        id_to_opcode_map[opcode_id] = instr->getOpcode();
      }

      int opcode_id = opcode_to_id_map[instr->getOpcode()];

      /* check all operands. For now, we ignore constant, TEX, predicates and
       * unified registers. We only report vector regisers */
      int src_oprd[MAX_SRC];
      int srcNum = 0;
      int dst_oprd = -1;
      int mem_oper_idx = -1;
      int num_mref = 0;
      uint64_t imm_value = 0;

      for (int i = 0; i < instr->getNumOperands(); ++i) {
        const InstrType::operand_t *op = instr->getOperand(i);
        if (op->type == InstrType::OperandType::MREF) {
          assert(srcNum < MAX_SRC);
          src_oprd[srcNum] = instr->getOperand(i)->u.mref.ra_num;
          srcNum++;
          mem_oper_idx++;
          num_mref++;
          // if(mem_oper_idx == 0){
          //   mem_oper_idx = 1; // loop control
          // }
        } else if (op->type == InstrType::OperandType::REG) {
          if (i == 0) {
            // find dst reg
            dst_oprd = instr->getOperand(0)->u.reg.num;
          } else {
            // find src regs
            assert(srcNum < MAX_SRC);
            src_oprd[srcNum] = instr->getOperand(i)->u.reg.num;
            srcNum++;
          }
        }
        // Add immediate value for DEPBAR instruction
        else if (op->type == InstrType::OperandType::IMM_UINT64) {
          imm_value = instr->getOperand(i)->u.imm_uint64.value;
        }
      }

      do {
        /* insert call to the instrumentation function with its
         * arguments */
        nvbit_insert_call(instr, "instrument_inst", IPOINT_BEFORE);

        /* pass predicate value */
        nvbit_add_call_arg_guard_pred_val(instr);

        /* send opcode and pc */
        nvbit_add_call_arg_const_val32(instr, opcode_id);
        nvbit_add_call_arg_const_val32(instr, (int)instr->getOffset());

        /* mem addresses info */
        if (mem_oper_idx >= 0) {
          nvbit_add_call_arg_const_val32(instr, 1);
          assert(num_mref <= 2);
          if (num_mref == 2) { // LDGSTS
            nvbit_add_call_arg_mref_addr64(instr, 1 - mem_oper_idx);
          } else {
            nvbit_add_call_arg_mref_addr64(instr, mem_oper_idx);
          }
          nvbit_add_call_arg_const_val32(instr, (int)instr->getSize());
        } else {
          nvbit_add_call_arg_const_val32(instr, 0);
          nvbit_add_call_arg_const_val64(instr, static_cast<uint64_t>(-1));
          nvbit_add_call_arg_const_val32(instr, static_cast<uint32_t>(-1));
        }

        /* reg info */
        nvbit_add_call_arg_const_val32(instr, dst_oprd);
        for (int i = 0; i < srcNum; i++) {
          nvbit_add_call_arg_const_val32(instr, src_oprd[i]);
        }
        for (int i = srcNum; i < MAX_SRC; i++) {
          nvbit_add_call_arg_const_val32(instr, static_cast<uint32_t>(-1));
        }
        nvbit_add_call_arg_const_val32(instr, srcNum);

        /* immediate info */
        nvbit_add_call_arg_const_val64(instr, imm_value);

        /* add pointer to channel_dev and other counters*/
        nvbit_add_call_arg_const_val64(instr, (uint64_t)&channel_dev);
        nvbit_add_call_arg_const_val64(instr,
                                       (uint64_t)&total_dynamic_instr_counter);
        nvbit_add_call_arg_const_val64(
            instr, (uint64_t)&reported_dynamic_instr_counter);
        nvbit_add_call_arg_const_val64(instr, (uint64_t)&stop_report);
        /* Add Source code line number for current instr */
        nvbit_add_call_arg_const_val32(instr, (int)line_num);

        // Detect if this is a store operation and capture store data
        std::string opcode_str = instr->getOpcode();
        bool is_store_op = (opcode_str.find("ST") == 0 || 
                           opcode_str.find("STG") == 0 || 
                           opcode_str.find("STS") == 0 ||
                           opcode_str.find("STL") == 0 ||
                           opcode_str.find("ATOM") == 0 ||
                           opcode_str.find("RED") == 0);
        
        nvbit_add_call_arg_const_val32(instr, is_store_op ? 1 : 0);
        
        // For store operations, identify and capture the data registers
        std::vector<int> store_data_regs;
        store_data_type_t data_type = STORE_DATA_UNKNOWN;
        
        if (is_store_op && mem_oper_idx >= 0) {
          // Determine data type from opcode
          if (opcode_str.find(".F32") != std::string::npos || opcode_str.find(".32") != std::string::npos) {
            data_type = STORE_DATA_FLOAT32;
          } else if (opcode_str.find(".F64") != std::string::npos || opcode_str.find(".64") != std::string::npos) {
            data_type = STORE_DATA_FLOAT64;
          } else if (opcode_str.find(".U64") != std::string::npos || opcode_str.find(".S64") != std::string::npos) {
            data_type = STORE_DATA_INT64;
          } else if (opcode_str.find(".U32") != std::string::npos || opcode_str.find(".S32") != std::string::npos) {
            data_type = STORE_DATA_INT32;
          } else if (opcode_str.find(".U16") != std::string::npos || opcode_str.find(".S16") != std::string::npos) {
            data_type = STORE_DATA_INT16;
          } else if (opcode_str.find(".U8") != std::string::npos || opcode_str.find(".S8") != std::string::npos) {
            data_type = STORE_DATA_INT8;
          } else {
            // Default based on instruction size
            int size = instr->getSize();
            if (size == 8) data_type = STORE_DATA_INT64;
            else if (size == 4) data_type = STORE_DATA_INT32;
            else if (size == 2) data_type = STORE_DATA_INT16;
            else if (size == 1) data_type = STORE_DATA_INT8;
            else data_type = STORE_DATA_INT32; // fallback
          }
          
          // For store operations, the source registers (excluding address register) contain the data
          for (int i = 0; i < instr->getNumOperands(); ++i) {
            const InstrType::operand_t *op = instr->getOperand(i);
            if (op->type == InstrType::OperandType::REG) {
              // Skip the first operand if it's a destination (for atomic operations)
              // For regular stores, all register operands except the address register contain data
              if (i > 0 || (i == 0 && opcode_str.find("ATOM") != 0 && opcode_str.find("RED") != 0)) {
                // Check if this register is not the address register
                bool is_addr_reg = false;
                for (int j = 0; j < instr->getNumOperands(); ++j) {
                  const InstrType::operand_t *addr_op = instr->getOperand(j);
                  if (addr_op->type == InstrType::OperandType::MREF && 
                      addr_op->u.mref.ra_num == op->u.reg.num) {
                    is_addr_reg = true;
                    break;
                  }
                }
                if (!is_addr_reg) {
                  // For 64-bit data types, we need two consecutive registers
                  // For smaller types, we use one register
                  if (data_type == STORE_DATA_FLOAT64 || data_type == STORE_DATA_INT64) {
                    store_data_regs.push_back(op->u.reg.num);     // Lower 32 bits
                    store_data_regs.push_back(op->u.reg.num + 1); // Upper 32 bits
                  } else {
                    store_data_regs.push_back(op->u.reg.num);
                  }
                }
              }
            }
          }
        }
        
        // Add data type
        nvbit_add_call_arg_const_val32(instr, (int32_t)data_type);
        
        // Add number of store data registers
        nvbit_add_call_arg_const_val32(instr, store_data_regs.size());
        
        // Add store data register values based on data type
        for (int reg_num : store_data_regs) {
          // The boolean parameter indicates this is a variadic argument, not the data type
          // nvbit always captures register values as uint32_t regardless of actual content
          nvbit_add_call_arg_reg_val(instr, reg_num, true);  // true = variadic argument
        }

        mem_oper_idx--;
      } while (mem_oper_idx >= 0);

      cnt++;
    }
  }
}

__global__ void flush_channel() {
  /* push memory access with negative cta id to communicate the kernel is
   * completed */
  inst_trace_t ma;
  ma.cta_id_x = -1;
  channel_dev.push(&ma, sizeof(inst_trace_t));

  /* flush channel */
  channel_dev.flush();
}

static void enter_kernel_launch(hipCtx_t ctx, hipFunction_t func,
                                nvbit_api_cuda_t cbid, void *params,
                                bool stream_capture = false,
                                bool build_graph = false) {
  // no need to sync during stream capture or manual graph build, since no
  // kernel is actually launched.
  if (!stream_capture && !build_graph) {
    /* Make sure GPU is idle */
    hipDeviceSynchronize();
    assert(hipGetLastError() == hipSuccess);
  }

  // Mark if the kernel should be traced
  std::string func_name = std::string(nvbit_get_func_name(ctx, func, true));
  if (active_from_start && should_trace_kernel(ctx_kernelid[ctx], func_name))
    active_region = true;

  // Terminate tracing if the limit number of kernels is reached
  if (terminate_after_limit_number_of_kernels_reached &&
    g_max_kernel_id != 0 &&
      ctx_kernelid[ctx] > g_max_kernel_id) {
    exit(0);
  }

  // Get launch config for this kernel
  unsigned int gridDimX, gridDimY, gridDimZ;
  unsigned int blockDimX, blockDimY, blockDimZ;
  unsigned int sharedMemBytes;
  hipStream_t hStream;
  if (cbid == API_CUDA_cuLaunchKernelEx_ptsz ||
      cbid == API_CUDA_cuLaunchKernelEx) {
    cuLaunchKernelEx_params *p = (cuLaunchKernelEx_params *)params;
    gridDimX = p->config->gridDimX;
    gridDimY = p->config->gridDimY;
    gridDimZ = p->config->gridDimZ;
    blockDimX = p->config->blockDimX;
    blockDimY = p->config->blockDimY;
    blockDimZ = p->config->blockDimZ;
    sharedMemBytes = p->config->sharedMemBytes;
    hStream = p->config->hStream;
  } else {
    cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;
    gridDimX = p->gridDimX;
    gridDimY = p->gridDimY;
    gridDimZ = p->gridDimZ;
    blockDimX = p->blockDimX;
    blockDimY = p->blockDimY;
    blockDimZ = p->blockDimZ;
    sharedMemBytes = p->sharedMemBytes;
    hStream = p->hStream;
  }

  // Get the number of registers and shared memory size for the kernel
  int nregs;
  CUDA_SAFECALL(hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, func));

  int shmem_static_nbytes;
  CUDA_SAFECALL(hipFuncGetAttribute(&shmem_static_nbytes,
                                   HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, func));

  int binary_version;
  CUDA_SAFECALL(hipFuncGetAttribute(&binary_version,
                                   HIP_FUNC_ATTRIBUTE_BINARY_VERSION, func));

  // Instrument the kernel if needed
  instrument_function_if_needed(ctx, func);

  // Enable or disable tracing based on the active region
  if (active_region) {
    nvbit_enable_instrumented(ctx, func, true);
    stop_report = false;
  } else {
    nvbit_enable_instrumented(ctx, func, false);
    stop_report = true;
  }

  // Create the trace file per kernel
  char buffer[2048];
  std::string trace_filename = user_folder + "/traces/";
  sprintf(buffer, "%s/kernel-%d-ctx_0x%lx.trace", trace_filename.c_str(),
          ctx_kernelid[ctx], ctx);

  if (!stop_report) {
    if (!xz_compress_trace) {
      ctx_resultsFile[ctx] = fopen(buffer, "w");
      printf("Writing results to %s\n", buffer);
    } else {
      char cmd_buffer[1039];
      sprintf(cmd_buffer, "xz -1 -T0 > %s.xz", buffer);
      ctx_resultsFile[ctx] = popen(cmd_buffer, "w");
      printf("Writing results to %s.xz\n", buffer);
    }

    // Writing header information
    fprintf(ctx_resultsFile[ctx], "-kernel name = %s\n",
            nvbit_get_func_name(ctx, func, true));
    fprintf(ctx_resultsFile[ctx], "-kernel id = %d\n", ctx_kernelid[ctx]);
    fprintf(ctx_resultsFile[ctx], "-grid dim = (%d,%d,%d)\n", gridDimX,
            gridDimY, gridDimZ);
    fprintf(ctx_resultsFile[ctx], "-block dim = (%d,%d,%d)\n", blockDimX,
            blockDimY, blockDimZ);
    fprintf(ctx_resultsFile[ctx], "-shmem = %d\n",
            shmem_static_nbytes + sharedMemBytes);
    fprintf(ctx_resultsFile[ctx], "-nregs = %d\n", nregs);
    fprintf(ctx_resultsFile[ctx], "-binary version = %d\n", binary_version);
    fprintf(ctx_resultsFile[ctx], "-cuda stream id = %lu\n", (uint64_t)hStream);
    fprintf(ctx_resultsFile[ctx], "-shmem base_addr = 0x%016lx\n",
            (uint64_t)nvbit_get_shmem_base_addr(ctx));
    fprintf(ctx_resultsFile[ctx], "-local mem base_addr = 0x%016lx\n",
            (uint64_t)nvbit_get_local_mem_base_addr(ctx));
    fprintf(ctx_resultsFile[ctx], "-nvbit version = %s\n", NVBIT_VERSION);
    fprintf(ctx_resultsFile[ctx], "-accelsim tracer version = %s\n",
            TRACER_VERSION);
    fprintf(ctx_resultsFile[ctx], "-enable lineinfo = %d\n", lineinfo);
    fprintf(ctx_resultsFile[ctx], "\n");

    fprintf(ctx_resultsFile[ctx],
            "#traces format = [line_num] PC mask dest_num [reg_dests] "
            "opcode src_num "
            "[reg_srcs] mem_width [adrrescompress?] [mem_addresses] "
            "immediate [STORE_DATA num_regs [REGx:Ty:value ...]]\n");
    fprintf(ctx_resultsFile[ctx], "\n");
  }

  kernelsFile = fopen(ctx_kernelslist[ctx].c_str(), "a");
  // This will be a relative path to the traces file

  sprintf(buffer, "kernel-%d-ctx_0x%lx.trace%s", ctx_kernelid[ctx], ctx,
          xz_compress_trace ? ".xz" : "");
  if (!stop_report) {
    fprintf(kernelsFile, buffer);
    fprintf(kernelsFile, "\n");
  }
  fclose(kernelsFile);

  statsFile = fopen(ctx_stats_location[ctx].c_str(), "a");
  unsigned blocks = gridDimX * gridDimY * gridDimZ;
  unsigned threads = blockDimX * blockDimY * blockDimZ;

  fprintf(statsFile, "%s, %s, %d, %d, %d, %d, %d, %d, %d, %d, ", buffer,
          nvbit_get_func_name(ctx, func, true), gridDimX, gridDimY, gridDimZ,
          blocks, blockDimX, blockDimY, blockDimZ, threads);

  fclose(statsFile);

  ctx_kernelid[ctx]++;
  recv_thread_receiving = true;
}

static void leave_kernel_launch(hipCtx_t ctx, hipFunction_t func) {
  /* make sure current kernel is completed */
  hipDeviceSynchronize();
  assert(hipGetLastError() == hipSuccess);

  /* make sure we prevent re-entry on the nvbit_callback when issuing
   * the flush_channel kernel */
  skip_flag = true;

  /* issue flush of channel so we are sure all the memory accesses
   * have been pushed */
  flush_channel<<<1, 1>>>();
  hipDeviceSynchronize();
  assert(hipGetLastError() == hipSuccess);

  /* unset the skip flag */
  skip_flag = false;

  /* wait here until the receiving thread has not finished with the
   * current kernel */
  while (recv_thread_receiving) {
    pthread_yield();
  }

  unsigned total_insts_per_kernel =
      total_dynamic_instr_counter - old_total_insts;
  old_total_insts = total_dynamic_instr_counter;

  unsigned reported_insts_per_kernel =
      reported_dynamic_instr_counter - old_total_reported_insts;
  old_total_reported_insts = reported_dynamic_instr_counter;

  statsFile = fopen(ctx_stats_location[ctx].c_str(), "a");
  fprintf(statsFile, "%d,%d", total_insts_per_kernel,
          reported_insts_per_kernel);
  fprintf(statsFile, "\n");
  fclose(statsFile);

  if (!stop_report) {
    if (!xz_compress_trace) {
      fclose(ctx_resultsFile[ctx]);
    } else {
      pclose(ctx_resultsFile[ctx]);
    }
  }

  std::string func_name = std::string(nvbit_get_func_name(ctx, func, true));
  if (active_from_start && !should_trace_kernel(ctx_kernelid[ctx], func_name))
    active_region = false;
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
  if (skip_flag)
    return;

  if (first_call == true) {
    first_call = false;
    std::string traces_folder = user_folder + "/traces";
    if (mkdir(traces_folder.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH) ==
        -1) {
      if (errno == EEXIST) {
        // alredy exists
      } else {
        // something else
        std::cout << "cannot create folder error:" << strerror(errno)
                  << std::endl;
        return;
      }
    }
    kernelsFile = fopen(ctx_kernelslist[ctx].c_str(), "w");
    statsFile = fopen(ctx_stats_location[ctx].c_str(), "w");
    fprintf(statsFile,
            "kernel id, kernel mangled name, grid_dimX, grid_dimY, grid_dimZ, "
            "#blocks, block_dimX, block_dimY, block_dimZ, #threads, "
            "total_insts, total_reported_insts\n");
    fclose(statsFile);
  }

  switch (cbid) {
  // We start with recording kernel launch events
  // Identify all the possible CUDA launch events without stream
  // parameters, they will not get involved with cuda graph
  case API_CUDA_cuLaunch:
  case API_CUDA_cuLaunchGrid: {
    cuLaunch_params *p = (cuLaunch_params *)params;
    hipFunction_t func = p->f;
    if (!is_exit) {
      enter_kernel_launch(ctx, func, cbid, params, false, false);
    } else {
      leave_kernel_launch(ctx, func);
    }
  } break;
  // To support kernel launched by cuda graph (in addition to existing kernel
  // launche method), we need to do:
  //
  // 1. instrument kernels at hipGraphAddKernelNode event. This is for cases
  // that kernels are manually added to a cuda graph.
  // 2. distinguish captured kernels when kernels are recorded to a graph
  // using stream capture. hipStreamIsCapturing() tells us whether a stream
  // is capturiong.
  // 3. per-kernel instruction counters, since cuda graph can launch multiple
  // kernels at the same time.
  //
  // Three cases:
  //
  // 1. original kernel launch:
  //     1a. for any kernel launch without using a stream, we instrument it
  //     before it is launched, call hipDeviceSynchronize after it is
  //     launched and read the instruction counter of the kernel.
  //     1b. for any kernel launch using a stream, but the stream is not
  //     capturing, we do the same thing as 1a.
  //
  //  2. cuda graph using stream capturing: if a kernel is launched in a
  //  stream and the stream is capturing. We instrument the kernel before it
  //  is launched and do nothing after it is launched, because the kernel is
  //  not running until hipGraphLaunch. Instead, we issue a
  //  hipStreamSynchronize after hipGraphLaunch is done and reset the
  //  instruction counters, since a cloned graph might be launched afterwards.
  //
  //  3. cuda graph manual: we instrument the kernel added by
  //  hipGraphAddKernelNode and do the same thing for hipGraphLaunch as 2.
  //
  // The above method should handle most of cuda graph launch cases.
  // kernel launches with stream parameter, they can be used for cuda graph
  case API_CUDA_cuLaunchKernel_ptsz:
  case API_CUDA_cuLaunchKernel:
  case API_CUDA_cuLaunchCooperativeKernel:
  case API_CUDA_cuLaunchCooperativeKernel_ptsz:
  case API_CUDA_cuLaunchKernelEx:
  case API_CUDA_cuLaunchKernelEx_ptsz:
  case API_CUDA_cuLaunchGridAsync: {
    hipFunction_t func;
    hipStream_t hStream;

    if (cbid == API_CUDA_cuLaunchKernelEx_ptsz ||
        cbid == API_CUDA_cuLaunchKernelEx) {
      cuLaunchKernelEx_params *p = (cuLaunchKernelEx_params *)params;
      func = p->f;
      hStream = p->config->hStream;
    } else if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
               cbid == API_CUDA_cuLaunchKernel ||
               cbid == API_CUDA_cuLaunchCooperativeKernel_ptsz ||
               cbid == API_CUDA_cuLaunchCooperativeKernel) {
      cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;
      func = p->f;
      hStream = p->hStream;
    } else {
      cuLaunchGridAsync_params *p = (cuLaunchGridAsync_params *)params;
      func = p->f;
      hStream = p->hStream;
    }

    hipStreamCaptureStatus streamStatus;
    /* check if the stream is capturing, if yes, do not sync */
    CUDA_SAFECALL(hipStreamIsCapturing(hStream, &streamStatus));
    if (!is_exit) {
      bool stream_capture = (streamStatus == hipStreamCaptureStatusActive);
      enter_kernel_launch(ctx, func, cbid, params, stream_capture);
    } else {
      if (streamStatus != hipStreamCaptureStatusActive) {
        if (verbose >= 1) {
          printf("kernel %s not captured by cuda graph\n",
                 nvbit_get_func_name(ctx, func));
        }
        leave_kernel_launch(ctx, func);
      } else {
        if (verbose >= 1) {
          printf("kernel %s captured by cuda graph\n",
                 nvbit_get_func_name(ctx, func));
        }
      }
    }
  } break;
  case API_CUDA_cuGraphAddKernelNode: {
    cuGraphAddKernelNode_params *p = (cuGraphAddKernelNode_params *)params;
    hipFunction_t func = p->nodeParams->func;

    if (!is_exit) {
      // cuGraphAddKernelNode_params->nodeParams is the same as
      // cuLaunchKernel_params up to sharedMemBytes
      enter_kernel_launch(ctx, func, cbid, (void *)p->nodeParams, false, true);
    }
  } break;
  case API_CUDA_cuGraphLaunch: {
    // if we are exiting a cuda graph launch:
    // Wait until the graph is completed using
    // hipStreamSynchronize()
    if (is_exit) {
      cuGraphLaunch_params *p = (cuGraphLaunch_params *)params;

      CUDA_SAFECALL(hipStreamSynchronize(p->hStream));
      assert(hipGetLastError() == hipSuccess);
      /* push a flush channel kernel */
      flush_channel<<<1, 1, 0, p->hStream>>>();
      CUDA_SAFECALL(hipStreamSynchronize(p->hStream));
      assert(hipGetLastError() == hipSuccess);
    }

  } break;
  // Now we need to record cuda memcpy events
  case API_CUDA_cuMemcpyHtoD_v2: {
    if (!is_exit) {
      cuMemcpyHtoD_v2_params *p = (cuMemcpyHtoD_v2_params *)params;
      char buffer[1024];
      
      // Generate memory dump filename if enabled
      std::string memory_dump_filename = "";
      if (enable_memory_dump) {
        sprintf(buffer, "memcpy_htod_%d_ctx_0x%lx_addr_0x%016llx_size_%llu.bin", 
                ctx_memcpy_counter[ctx], ctx, p->dstDevice, p->ByteCount);
        memory_dump_filename = user_folder + "/traces/" + buffer;
        
        // Dump the host memory data to binary file
        FILE *dump_file = NULL;
        if (memory_dump_compress) {
          char cmd_buffer[2048];
          sprintf(cmd_buffer, "xz -1 -T0 > %s.xz", memory_dump_filename.c_str());
          dump_file = popen(cmd_buffer, "wb");
          if (verbose) {
            printf("Dumping compressed memory to %s.xz (size: %llu bytes)\n", 
                   memory_dump_filename.c_str(), p->ByteCount);
          }
        } else {
          dump_file = fopen(memory_dump_filename.c_str(), "wb");
          if (verbose) {
            printf("Dumping memory to %s (size: %llu bytes)\n", 
                   memory_dump_filename.c_str(), p->ByteCount);
          }
        }
        
        if (dump_file != NULL) {
          // Write the host memory data to the dump file
          size_t written = fwrite(p->srcHost, 1, p->ByteCount, dump_file);
          if (written != p->ByteCount) {
            printf("Warning: Only wrote %zu out of %llu bytes to memory dump file\n", 
                   written, p->ByteCount);
          }
          
          if (memory_dump_compress) {
            pclose(dump_file);
          } else {
            fclose(dump_file);
          }
        } else {
          printf("Error: Failed to create memory dump file %s\n", memory_dump_filename.c_str());
        }
      }
      
      // Write to kernelslist
      kernelsFile = fopen(ctx_kernelslist[ctx].c_str(), "a");
      if (enable_memory_dump) {
        // Include memory dump filename in kernelslist entry
        sprintf(buffer, "MemcpyHtoD,0x%016llx,%llu,%s%s", 
                p->dstDevice, p->ByteCount, 
                memory_dump_filename.substr(memory_dump_filename.find_last_of('/') + 1).c_str(),
                memory_dump_compress ? ".xz" : "");
      } else {
        // Original format without memory dump
        sprintf(buffer, "MemcpyHtoD,0x%016llx,%llu", p->dstDevice, p->ByteCount);
      }
      fprintf(kernelsFile, buffer);
      fprintf(kernelsFile, "\n");
      fclose(kernelsFile);
      
      // Increment memory copy counter
      ctx_memcpy_counter[ctx]++;
    }
  } break;
  // For cuProfiler, we need to set the active region accordingly
  case API_CUDA_cuProfilerStart: {
    if (is_exit && !active_from_start) {
      active_region = true;
    }
  } break;
  case API_CUDA_cuProfilerStop: {
    if (is_exit && !active_from_start) {
      active_region = false;
    }
  } break;
  default:
    break;
  }
}

bool is_number(const std::string &s) {
  std::string::const_iterator it = s.begin();
  while (it != s.end() && std::isdigit(*it))
    ++it;
  return !s.empty() && it == s.end();
}

unsigned get_datawidth_from_opcode(const std::vector<std::string> &opcode) {
  for (unsigned i = 0; i < opcode.size(); ++i) {
    if (is_number(opcode[i])) {
      return (std::stoi(opcode[i], NULL) / 8);
    } else if (opcode[i][0] == 'U' && is_number(opcode[i].substr(1))) {
      // handle the U* case
      unsigned bits;
      sscanf(opcode[i].c_str(), "U%u", &bits);
      return bits / 8;
    }
  }

  return 4; // default is 4 bytes
}

bool check_opcode_contain(const std::vector<std::string> &opcode,
                          std::string param) {
  for (unsigned i = 0; i < opcode.size(); ++i)
    if (opcode[i] == param)
      return true;

  return false;
}

bool base_stride_compress(const uint64_t *addrs, const std::bitset<32> &mask,
                          uint64_t &base_addr, int &stride) {
  // calulcate the difference between addresses
  // write cosnsctive addresses with constant stride in a more
  // compressed way (i.e. start adress and stride)
  bool const_stride = true;
  bool first_bit1_found = false;
  bool last_bit1_found = false;

  for (int s = 0; s < 32; s++) {
    if (mask.test(s) && !first_bit1_found) {
      first_bit1_found = true;
      base_addr = addrs[s];
      if (s < 31 && mask.test(s + 1))
        stride = addrs[s + 1] - addrs[s];
      else {
        const_stride = false;
        break;
      }
    } else if (first_bit1_found && !last_bit1_found) {
      if (mask.test(s)) {
        if (stride != addrs[s] - addrs[s - 1]) {
          const_stride = false;
          break;
        }
      } else
        last_bit1_found = true;
    } else if (last_bit1_found) {
      if (mask.test(s)) {
        const_stride = false;
        break;
      }
    }
  }

  return const_stride;
}

void base_delta_compress(const uint64_t *addrs, const std::bitset<32> &mask,
                         uint64_t &base_addr, std::vector<long long> &deltas) {
  // save the delta from the previous address
  bool first_bit1_found = false;
  uint64_t last_address = 0;
  for (int s = 0; s < 32; s++) {
    if (mask.test(s) && !first_bit1_found) {
      base_addr = addrs[s];
      first_bit1_found = true;
      last_address = addrs[s];
    } else if (mask.test(s) && first_bit1_found) {
      deltas.push_back(addrs[s] - last_address);
      last_address = addrs[s];
    }
  }
}

void *recv_thread_fun(void *args) {
  hipCtx_t ctx = (hipCtx_t)args;
  char *recv_buffer = (char *)malloc(CHANNEL_SIZE);
  while (recv_thread_started) {
    uint32_t num_recv_bytes = 0;
    if (recv_thread_receiving &&
        (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) > 0) {
      uint32_t num_processed_bytes = 0;
      while (num_processed_bytes < num_recv_bytes) {
        inst_trace_t *ma = (inst_trace_t *)&recv_buffer[num_processed_bytes];

        /* when we get this cta_id_x it means the kernel has completed
         */
        if (ma->cta_id_x == -1) {
          recv_thread_receiving = false;
          break;
        }

        fprintf(ctx_resultsFile[ctx], "%d ", ma->cta_id_x);
        fprintf(ctx_resultsFile[ctx], "%d ", ma->cta_id_y);
        fprintf(ctx_resultsFile[ctx], "%d ", ma->cta_id_z);
        fprintf(ctx_resultsFile[ctx], "%d ", ma->warpid_tb);
        if (print_core_id) {
          fprintf(ctx_resultsFile[ctx], "%d ", ma->sm_id);
          fprintf(ctx_resultsFile[ctx], "%d ", ma->warpid_sm);
        }
        if (lineinfo) {
          fprintf(ctx_resultsFile[ctx], "%d ", ma->line_num);
        }
        fprintf(ctx_resultsFile[ctx], "%04x ", ma->vpc); // Print the virtual PC
        fprintf(ctx_resultsFile[ctx], "%08x ",
                ma->active_mask & ma->predicate_mask);
        if (ma->GPRDst >= 0) {
          fprintf(ctx_resultsFile[ctx], "1 ");
          fprintf(ctx_resultsFile[ctx], "R%d ", ma->GPRDst);
        } else
          fprintf(ctx_resultsFile[ctx], "0 ");

        // Print the opcode.
        fprintf(ctx_resultsFile[ctx], "%s ",
                id_to_opcode_map[ma->opcode_id].c_str());
        unsigned src_count = 0;
        for (int s = 0; s < MAX_SRC; s++) // GPR srcs count.
          if (ma->GPRSrcs[s] >= 0)
            src_count++;
        fprintf(ctx_resultsFile[ctx], "%d ", src_count);

        for (int s = 0; s < MAX_SRC; s++) // GPR srcs.
          if (ma->GPRSrcs[s] >= 0)
            fprintf(ctx_resultsFile[ctx], "R%d ", ma->GPRSrcs[s]);

        // print addresses
        std::bitset<32> mask(ma->active_mask & ma->predicate_mask);
        if (ma->is_mem) {
          std::istringstream iss(id_to_opcode_map[ma->opcode_id]);
          std::vector<std::string> tokens;
          std::string token;
          while (std::getline(iss, token, '.')) {
            if (!token.empty())
              tokens.push_back(token);
          }
          fprintf(ctx_resultsFile[ctx], "%d ",
                  get_datawidth_from_opcode(tokens));

          bool base_stride_success = false;
          uint64_t base_addr = 0;
          int stride = 0;
          std::vector<long long> deltas;

          if (enable_compress) {
            // try base+stride format
            base_stride_success =
                base_stride_compress(ma->addrs, mask, base_addr, stride);
            if (!base_stride_success) {
              // if base+stride fails, try base+delta format
              base_delta_compress(ma->addrs, mask, base_addr, deltas);
            }
          }

          if (base_stride_success && enable_compress) {
            // base + stride format
            fprintf(ctx_resultsFile[ctx], "%u 0x%llx %d ",
                    address_format::base_stride, base_addr, stride);
          } else if (!base_stride_success && enable_compress) {
            // base + delta format
            fprintf(ctx_resultsFile[ctx], "%u 0x%llx ",
                    address_format::base_delta, base_addr);
            for (int s = 0; s < deltas.size(); s++) {
              fprintf(ctx_resultsFile[ctx], "%lld ", deltas[s]);
            }
          } else {
            // list all the addresses
            fprintf(ctx_resultsFile[ctx], "%u ", address_format::list_all);
            for (int s = 0; s < 32; s++) {
              if (mask.test(s))
                fprintf(ctx_resultsFile[ctx], "0x%016lx ", ma->addrs[s]);
            }
          }
        } else {
          fprintf(ctx_resultsFile[ctx], "0 ");
        }

        // Print the immediate
        fprintf(ctx_resultsFile[ctx], "%d ", ma->imm);

        // Print store data if this is a store operation
        if (ma->is_store && ma->num_store_data_regs > 0) {
          const char* type_names[] = {"UNK", "I8", "I16", "I32", "I64", "F32", "F64"};
          fprintf(ctx_resultsFile[ctx], "STORE_DATA %s %d ", 
                  type_names[ma->store_data_type], ma->num_store_data_regs);
          
          for (int reg_idx = 0; reg_idx < ma->num_store_data_regs; reg_idx++) {
            fprintf(ctx_resultsFile[ctx], "REG%d:", reg_idx);
            for (int tid = 0; tid < 32; tid++) {
              if (mask.test(tid)) {
                // Format output based on data type
                switch (ma->store_data_type) {
                  case STORE_DATA_FLOAT32: {
                    uint32_t int_val = (uint32_t)ma->store_data[tid][reg_idx];
                    float float_val = *(float*)&int_val;
                    fprintf(ctx_resultsFile[ctx], "T%d:%g ", tid, float_val);
                    break;
                  }
                  case STORE_DATA_FLOAT64: {
                    uint64_t int_val = ma->store_data[tid][reg_idx];
                    double double_val = *(double*)&int_val;
                    fprintf(ctx_resultsFile[ctx], "T%d:%g ", tid, double_val);
                    break;
                  }
                  case STORE_DATA_INT64:
                    fprintf(ctx_resultsFile[ctx], "T%d:0x%016llx ", tid, ma->store_data[tid][reg_idx]);
                    break;
                  case STORE_DATA_INT32:
                    fprintf(ctx_resultsFile[ctx], "T%d:0x%08x ", tid, (uint32_t)ma->store_data[tid][reg_idx]);
                    break;
                  case STORE_DATA_INT16:
                    fprintf(ctx_resultsFile[ctx], "T%d:0x%04x ", tid, (uint16_t)ma->store_data[tid][reg_idx]);
                    break;
                  case STORE_DATA_INT8:
                    fprintf(ctx_resultsFile[ctx], "T%d:0x%02x ", tid, (uint8_t)ma->store_data[tid][reg_idx]);
                    break;
                  default:
                    fprintf(ctx_resultsFile[ctx], "T%d:0x%016llx ", tid, ma->store_data[tid][reg_idx]);
                    break;
                }
              }
            }
            fprintf(ctx_resultsFile[ctx], " ");
          }
        }

        fprintf(ctx_resultsFile[ctx], "\n");

        num_processed_bytes += sizeof(inst_trace_t);
      }
    }
  }
  free(recv_buffer);
  return NULL;
}

void nvbit_tool_init(hipCtx_t ctx) {
  recv_thread_started = true;
  channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
  pthread_create(&recv_thread, NULL, recv_thread_fun, ctx);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
  if (recv_thread_started) {
    recv_thread_started = false;
    pthread_join(recv_thread, NULL);
  }
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
  // Everytime we init a context, add the foldername and kernelid to the set
  char buffer[2048];
  sprintf(buffer, "kernelslist_ctx_0x%lx", ctx);
  std::string tmp_kernelslist = user_folder + "/traces/" + buffer;
  ctx_kernelslist[ctx] = tmp_kernelslist;
  sprintf(buffer, "stats_ctx_0x%lx", ctx);
  std::string tmp_stats = user_folder + "/traces/" + buffer;
  ctx_stats_location[ctx] = tmp_stats;
  ctx_kernelid[ctx] = 1;
  ctx_memcpy_counter[ctx] = 0;
}